#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include "bitset"

#include "opencv2/core.hpp"
#include "opencv2/highgui.hpp"
#include <stdio.h>

#include <stdint.h>

using namespace std;
using namespace cv;

typedef unsigned char uchar;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort =
		true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
				line);
		if (abort)
			exit(code);
	}
}

__device__
inline int clamp(int x, int min_, int max_) {
	if (x < min_)
		x = min_;
	else if (x > max_)
		x = max_;
	return x;
}

__device__ int clamp_2d_position(int w, int h, int gx, int gy) {
	if (gx >= w) {
		return -1;
	}
	if (gy >= h)
		return -1;

	int pos = gy * w + gx;
	if (pos >= w * h) {
		return -1;
	}
	return pos;
}

__global__
void cencus(const uchar* in, uint32_t* out, int w, int h) {
	int gx = blockIdx.x * blockDim.x + threadIdx.x;
	int gy = blockIdx.y * blockDim.y + threadIdx.y;

	int pos = clamp_2d_position(w, h, gx, gy);
	if (pos < 0) {
		return;
	}

	int k1 = 3;
	int hk1 = k1 >> 1;

	uint32_t res = 0;
	uint32_t mask = 0x01 << (k1 * k1 - 1);

	uchar I = in[pos];
	for (int dx = -hk1; dx < hk1; ++dx) {
		for (int dy = -hk1; dy < hk1; ++dy) {
			int x = clamp(gx + dx, 0, w);
			int y = clamp(gy + dy, 0, h);

			uchar Ie = in[y * w + x];
			if (I < Ie) {
				res |= mask;
			} else {
				res &= ~(mask);
			}
			mask >>= 1;
		}
	}

	out[pos] = res;
}

// [0, 32]
#define MAX_DISP 32
#define WS 7

__global__ void sbm_census(uint32_t* i0, uint32_t* i1, int w, int h,
		short* d_disp) {
	int gx = blockIdx.x * blockDim.x + threadIdx.x;
	int gy = blockIdx.y * blockDim.y + threadIdx.y;

	int pos = clamp_2d_position(w, h, gx, gy);
	if (pos < 0) {
		return;
	}

	// fixme: можно кстати искать на меньшем диапазоне, а не выкидывать
	if (gx < MAX_DISP) {
		return;
	}

	short* energy[MAX_DISP];
	uint32_t i0_bs = i0[pos];

	// for ...

}

void census(uchar* d_img, uint32_t* d_img_census, int w, int h) {
	const dim3 wg(w / 16 + 1, h / 16 + 1);
	const dim3 bs(16, 16);
	// measure
	// https://devblogs.nvidia.com/parallelforall/how-implement-performance-metrics-cuda-cc/
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	// Perform SAXPY on 1M elements

	cencus<<<wg, bs>>>(d_img, d_img_census, w, h);

	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << "elapsed:" << milliseconds / 1e3 << endl;
}

struct gpu_img_triple_t {

	gpu_img_triple_t(int N, uchar* h_i0, uchar* h_i1) {
		d_i0 = 0;
		d_i0_census = 0;
		d_i1 = 0;
		d_i1_census = 0;

		// im0
		gpuErrchk(hipMalloc(&d_i0, N * sizeof(uchar)));
		gpuErrchk(hipMalloc(&d_i0_census, N * sizeof(uint32_t)));

		gpuErrchk(
				hipMemcpy(d_i0, h_i0, N * sizeof(uchar),
						hipMemcpyHostToDevice));

		// im1
		gpuErrchk(hipMalloc(&d_i1, N * sizeof(uchar)));
		gpuErrchk(hipMalloc(&d_i1_census, N * sizeof(uint32_t)));

		gpuErrchk(
				hipMemcpy(d_i1, h_i1, N * sizeof(uchar),
						hipMemcpyHostToDevice));

		// disp
		gpuErrchk(hipMalloc(&d_disp_i16, N * sizeof(short)));
	}

	~gpu_img_triple_t() {
		gpuErrchk(hipFree(d_i0));
		gpuErrchk(hipFree(d_i0_census));
		gpuErrchk(hipFree(d_i1));
		gpuErrchk(hipFree(d_i1_census));
		gpuErrchk(hipFree(d_disp_i16));
	}

	uchar *d_i0;
	uint32_t *d_i0_census;
	uchar *d_i1;
	uint32_t *d_i1_census;

	short* d_disp_i16;
};

int main(void) {

	Mat im0 = imread("im0.png", 0);
	Mat im1 = imread("im1.png", 0);

	int w = im0.cols;
	int h = im0.rows;
	int N = w * h;

	gpu_img_triple_t gpair(N, im0.data, im1.data);

	// census
	vector<uint32_t> h_i0_census(N);
	vector<uint32_t> h_i1_census(N);
	census(gpair.d_i0, gpair.d_i0_census, w, h);

	gpuErrchk(
			hipMemcpy(&h_i0_census[0], gpair.d_i0_census, N * sizeof(uint32_t),
					hipMemcpyDeviceToHost));

	// https://stackoverflow.com/questions/14581806/can-not-use-cv-32uc1
	Mat A = Mat(h, w, CV_32S, &h_i0_census[0]);

	Mat B;
	A.convertTo(B, CV_8U);
	imwrite("out0.png", B);

	census(gpair.d_i1, gpair.d_i1_census, w, h);
	gpuErrchk(
			hipMemcpy(&h_i1_census[0], gpair.d_i1_census, N * sizeof(uint32_t),
					hipMemcpyDeviceToHost));

	A = Mat(h, w, CV_32S, &h_i1_census[0]);

	A.convertTo(B, CV_8U);
	imwrite("out1.png", B);

	// matching
	{
		const dim3 wg(w / 16 + 1, h / 16 + 1);
		const dim3 bs(16, 16);
		// measure
		// https://devblogs.nvidia.com/parallelforall/how-implement-performance-metrics-cuda-cc/
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);

		sbm_census<<<wg, bs>>>(gpair.d_i0_census, gpair.d_i1_census, w, h,
				gpair.d_disp_i16);

		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		hipEventRecord(stop);

		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);

		cout << "elapsed:" << milliseconds / 1e3 << endl;
	}

}

